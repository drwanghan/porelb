#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <time.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include "mytype.h"
#include "simgpu.cu"

int main(int argc, char* argv[])
{
    float elpsd_time;
    clock_t time_begin, time_end; 

    SIMGPU sim;
    int it=0;
    if(argc != 4){
        fprintf(stderr, "Useage: ./simgpu itMax n|c GPU_ID\n");
        exit(-2);
    }

    setPara(&sim, argc, argv);
    allocMemroy(&sim);
    buildDataArray(&it, &sim);
    /*printf("It = %d \n", it);*/
    sim.itMax += it;

    int GX;
    int GXmax = 256;
    GX = (sim.num_node + BX-1)/BX;
    GX = (GX>GXmax) ? GXmax : GX;
    dim3 grid(GX, (sim.num_node + GX*BX-1)/(GX*BX), 1);  //grid layout for LBCollProp kernel

    hipSetDevice(atoi(argv[3]));
    printf("Using GPU %d\n", atoi(argv[3]));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    time_begin = clock();
    //----------------------8<-----------------------------------------------
    /*K_old = 1.0f;*/
    MY_DTYPE *fOld, *fNew;
    MY_DTYPE *gOld, *gNew;
    for(; it<sim.itMax; it++) {
        if(it%2==0) {
            fOld = sim.f0_d;
            fNew = sim.f1_d;
            gOld = sim.g0_d;
            gNew = sim.g1_d;
        } else {
            fOld = sim.f1_d;
            fNew = sim.f0_d;
            gOld = sim.g1_d;
            gNew = sim.g0_d;
        }

        LBCollProp<<<grid,  BX>>>(
                it,
                sim.num_node,
                sim.omegaF, sim.omegaG, sim.rhoS,
                sim.G_fg, sim.G_fs, sim.G_gs,
                fOld, fNew, gOld, gNew,
                sim.rhoF_d, sim.rhoG_d,
                sim.vxF_d, sim.vxG_d, sim.vyF_d, sim.vyG_d, sim.vzF_d, sim.vzG_d,
                sim.node_map_d);
        /*getLastCudaError("LBCollProp Launch failed!\n");*/

        LBUpdateMacro<<<grid, BX>>>(
                sim.num_node,
                sim.rhoF_d, sim.rhoG_d,
                sim.vxF_d, sim.vxG_d, sim.vyF_d, sim.vyG_d, sim.vzF_d, sim.vzG_d,
                fNew, gNew);
        /*getLastCudaError("LBUpdateMacro Launch failed!\n");*/

        if(it%1000==0) {
            copyBackData(&sim);
            /*outputData(&sim, it);*/
            printf("it : %8d, f11=% e, massError=% e\n", it, sim.f0_h[sim.num_node*12 + 1000], massError(&sim));
        }
    }
    time_end = clock();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elpsd_time, start, stop);
    printf("Time          : %f seconds \n", elpsd_time/1000.0);
    printf("Speed(CPU API): %f MLUPS\n", (float)(sim.num_node)*(float)(sim.itMax)/1000000.0/((float)(time_end-time_begin)/CLOCKS_PER_SEC));
    printf("Speed(CUDA API): %fMLUPS\n", (float)(sim.num_node)*(float)(sim.itMax)/1000000.0/elpsd_time*1000.0);

    copyBackData(&sim);
    correctVel(&sim);
    outputData(&sim, it);
    saveLoadRecovery(&it, &sim, 's');
    saveTecplot(&sim);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    freeMemrory(&sim);
    return 0;
}
