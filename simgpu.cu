#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "simgpu.h"
#include "node.h"
#include "lb.h"
#include "defs.h"
#include "mytype.h"
/*#include "hip/hip_runtime_api.h"*/
/*#include "helper_functions.h"*/
#include "TECIO.h"

/*Block size*/
#define BX 128
#define BY 1
#define BZ 1

const int e_h[Q][3] = {
    { 0,  0,  0},//0

    { 1,  0,  0},//1 
    {-1,  0,  0},//2
    { 0,  1,  0},//3
    { 0, -1,  0},//4
    { 0,  0,  1},//5
    { 0,  0, -1},//6

    { 0,  1,  1},
    { 0, -1, -1},
    { 0, -1,  1},
    { 0,  1, -1},

    {-1,  0, -1},
    { 1,  0,  1},
    {-1,  0,  1},
    { 1,  0, -1},

    {-1,  1,  0},
    { 1, -1,  0},
    {-1, -1,  0},
    { 1,  1,  0}
};
const MY_DTYPE w_h[Q] = {
    1.0/3, 
    1.0/18, 1.0/18, 1.0/18, 1.0/18, 1.0/18, 1.0/18,
    1.0/36, 1.0/36, 1.0/36, 1.0/36,
    1.0/36, 1.0/36, 1.0/36, 1.0/36, 
    1.0/36, 1.0/36, 1.0/36, 1.0/36
};

__constant__ int e_d[Q][3] = {
    { 0,  0,  0},//0

    { 1,  0,  0},//1 
    {-1,  0,  0},//2
    { 0,  1,  0},//3
    { 0, -1,  0},//4
    { 0,  0,  1},//5
    { 0,  0, -1},//6

    { 0,  1,  1},
    { 0, -1, -1},
    { 0, -1,  1},
    { 0,  1, -1},

    {-1,  0, -1},
    { 1,  0,  1},
    {-1,  0,  1},
    { 1,  0, -1},

    {-1,  1,  0},
    { 1, -1,  0},
    {-1, -1,  0},
    { 1,  1,  0}
};
__constant__ MY_DTYPE w_d[Q] = {
    1.0/3, 
    1.0/18, 1.0/18, 1.0/18, 1.0/18, 1.0/18, 1.0/18,
    1.0/36, 1.0/36, 1.0/36, 1.0/36,
    1.0/36, 1.0/36, 1.0/36, 1.0/36, 
    1.0/36, 1.0/36, 1.0/36, 1.0/36
};


void setPara(SIMGPU *simgpu_,  int argc,  char* argv[])
{
    simgpu_->nuF = 1.0/6;
    simgpu_->nuG = 1.0/6;
    
    simgpu_->tauF = 3.0*simgpu_->nuF + 0.50;
    simgpu_->tauG = 3.0*simgpu_->nuG + 0.50;

    printf("tauF = %f, tauG =%f\n", simgpu_->tauF,  simgpu_->tauG);
    simgpu_->omegaF = 1.0/simgpu_->tauF;
    simgpu_->omegaG = 1.0/simgpu_->tauG;

    simgpu_->rho_f = 1.0;
    simgpu_->rho_g = 1.0;
    simgpu_->rhoS = 1.0;

    simgpu_->G_fg =  0.200;
    simgpu_->G_fs =  0.030;
    simgpu_->G_gs =  -0.030;

    simgpu_->dt = 1.00;
    simgpu_->dx = 1.00;
    simgpu_->Fx = 1.0e-7;

    sprintf(simgpu_->node_map_filename, "%s", "V.bin");

    if(argc !=4){
        fprintf(stderr, "Argc error\n");
        exit(-1);
    }
    simgpu_->itMax = atoi(argv[1]);
    simgpu_->NewOrContinue = argv[2][0];
    //////////////////////////////////////
    printf("G_fg = %f\n", simgpu_->G_fg);
}

void allocMemroy(SIMGPU *simgpu_)
{
    FILE *fp_map;
    if((fp_map = fopen(simgpu_->node_map_filename, "rb")) == NULL)
    {
        fprintf(stderr, "Node map file openning error!\n");
        exit(-1);
    }
    fread(&simgpu_->num_node, sizeof(int), 1, fp_map);
    printf("num_node = %d\n", simgpu_->num_node);
    int num_p;
    fread(&num_p, sizeof(int), 1, fp_map);
    fread(&simgpu_->nx, sizeof(int), 1, fp_map);
    fread(&simgpu_->ny, sizeof(int), 1, fp_map);
    fread(&simgpu_->nz, sizeof(int), 1, fp_map);
    fseek(fp_map, num_p*sizeof(int), SEEK_CUR);
    simgpu_->L = (MY_DTYPE)simgpu_->nx;
    fclose(fp_map);


    //allocating memory at host
    simgpu_->f0_h = (MY_DTYPE *)calloc(Q*simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->g0_h = (MY_DTYPE *)calloc(Q*simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->rhoF_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->rhoG_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vxF_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vyF_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vzF_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vxG_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vyG_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vzG_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));

    simgpu_->vx_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vy_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));
    simgpu_->vz_h = (MY_DTYPE *)calloc(simgpu_->num_node, sizeof(MY_DTYPE));

    simgpu_->node_map_h = (unsigned int *)calloc(Q*simgpu_->num_node, sizeof(unsigned int));
    simgpu_->n_to_XYZ = (unsigned short *)calloc(3*simgpu_->num_node, sizeof(unsigned short));
    if(!( simgpu_->node_map_h && simgpu_->n_to_XYZ 
                && simgpu_->g0_h && simgpu_->f0_h
                && simgpu_->rhoF_h && simgpu_->rhoG_h )) {
        fprintf(stderr, "CPU memory allocating error\n");
        exit(-1);
    }

    //allocating memory at device
    hipMalloc((void **)&simgpu_->node_map_d, simgpu_->num_node*Q*sizeof(unsigned int));

    hipMalloc((void **)&simgpu_->f0_d, simgpu_->num_node*Q*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->f1_d, simgpu_->num_node*Q*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->g0_d, simgpu_->num_node*Q*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->g1_d, simgpu_->num_node*Q*sizeof(MY_DTYPE));


    hipMalloc((void **)&simgpu_->rhoF_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->rhoG_d, simgpu_->num_node*sizeof(MY_DTYPE));

    hipMalloc((void **)&simgpu_->vxF_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->vyF_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->vzF_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->vxG_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->vyG_d, simgpu_->num_node*sizeof(MY_DTYPE));
    hipMalloc((void **)&simgpu_->vzG_d, simgpu_->num_node*sizeof(MY_DTYPE));
}

void freeMemrory(SIMGPU *simgpu_)
{
    free(simgpu_->node_map_h); 
    free(simgpu_->n_to_XYZ); 

    free(simgpu_->f0_h); 
    free(simgpu_->g0_h); 
    free(simgpu_->rhoF_h);
    free(simgpu_->rhoG_h);
    free(simgpu_->vxF_h);
    free(simgpu_->vyF_h);
    free(simgpu_->vzF_h);
    free(simgpu_->vxG_h);
    free(simgpu_->vyG_h);
    free(simgpu_->vzG_h);
    free(simgpu_->vx_h);
    free(simgpu_->vy_h);
    free(simgpu_->vz_h);

    hipFree(simgpu_->node_map_d); 

    hipFree(simgpu_->f0_d);
    hipFree(simgpu_->f1_d);
    hipFree(simgpu_->g0_d);
    hipFree(simgpu_->g1_d);

    hipFree(simgpu_->rhoF_d);
    hipFree(simgpu_->rhoG_d);

    hipFree(simgpu_->vxF_d);
    hipFree(simgpu_->vyF_d);
    hipFree(simgpu_->vzF_d);
    hipFree(simgpu_->vxG_d);
    hipFree(simgpu_->vyG_d);
    hipFree(simgpu_->vzG_d);
}

void buildDataArray(int *it, SIMGPU *simgpu_)
{
    int num_node = simgpu_->num_node;
    int i,k;
    NODE_INFO node_info_tmp;
    FILE *fp_map;
    if((fp_map = fopen(simgpu_->node_map_filename, "rb")) == NULL) {
        fprintf(stderr, "Node map file openning error!\n");
        exit(-1);
    }
    fseek(fp_map, sizeof(int), SEEK_SET);
    int num_p;
    fread(&num_p, sizeof(int), 1, fp_map);
    fseek(fp_map, 3*sizeof(int), SEEK_CUR);
    fseek(fp_map, num_p*sizeof(int), SEEK_CUR);
    printf("num_p = %d\n", num_p);

    for(k=0; k<num_node; k++) {
        fread(&node_info_tmp,sizeof(NODE_INFO), 1 , fp_map);

        simgpu_->n_to_XYZ[0*num_node + k] = node_info_tmp.x;
        simgpu_->n_to_XYZ[1*num_node + k] = node_info_tmp.y;
        simgpu_->n_to_XYZ[2*num_node + k] = node_info_tmp.z;

        for(i=0;i<Q;i++) {
            if(i>0){
                if(node_info_tmp.nb_info[i-1].node_type == TYPE_SOLID){ //big bug here
                    simgpu_->node_map_h[i*num_node + k] = k + re[i]*num_node;
                }
                else 
                    simgpu_->node_map_h[i*num_node + k] = node_info_tmp.nb_info[i-1].node_id + i*num_node; //big bug here
            }
        }
    }
    fclose(fp_map);

    if(simgpu_->NewOrContinue == 'n')
        initDataArray(simgpu_);
    else
        saveLoadRecovery(it, simgpu_, 'l');

    hipMemcpy(simgpu_->node_map_d, simgpu_->node_map_h, num_node*Q*sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(simgpu_->f0_d, simgpu_->f0_h, num_node*Q*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->g0_d, simgpu_->g0_h, num_node*Q*sizeof(MY_DTYPE), hipMemcpyHostToDevice);

    hipMemcpy(simgpu_->rhoF_d, simgpu_->rhoF_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->rhoG_d, simgpu_->rhoG_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vxF_d, simgpu_->vxF_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vyF_d, simgpu_->vyF_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vzF_d, simgpu_->vzF_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vxG_d, simgpu_->vxG_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vyG_d, simgpu_->vyG_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
    hipMemcpy(simgpu_->vzG_d, simgpu_->vzG_h, num_node*sizeof(MY_DTYPE), hipMemcpyHostToDevice);
}

void initDataArray(SIMGPU *simgpu_)
{
    FILE *fp_rhoF, *fp_rhoG;
    fp_rhoF = fopen("rhoF.bin", "rb");
    fp_rhoG = fopen("rhoG.bin", "rb");
    if(!(fp_rhoG && fp_rhoF))
    {
        fprintf(stderr, "Rho file openning error!, simgpu.cu\n");
        exit(-1);
    }  
    int nx, ny, nz, num_node;
    unsigned short *n_to_XYZ;
    nx = simgpu_->nx;
    ny = simgpu_->ny;
    nz = simgpu_->nz;
    num_node = simgpu_->num_node;
    n_to_XYZ = simgpu_->n_to_XYZ;

    MY_DTYPE *rhoF, *rhoG;

    rhoF = (MY_DTYPE *)malloc(sizeof(MY_DTYPE)*nx*ny*nz);
    rhoG = (MY_DTYPE *)malloc(sizeof(MY_DTYPE)*nx*ny*nz);
    if(!(rhoF && rhoG)){
        fprintf(stderr, "Memory allocating error, simgpu.c\n");
        exit(-1);
    }

    if((fread(rhoF, sizeof(MY_DTYPE), nx*ny*nz, fp_rhoF)) != nx*ny*nz)
    {
        fprintf(stderr, "Rho file reading error!\n");
        exit(-1);
    }

    if((fread(rhoG, sizeof(MY_DTYPE), nx*ny*nz, fp_rhoG)) != nx*ny*nz)
    {
        fprintf(stderr, "Rho file reading error!\n");
        exit(-1);
    }

    fclose(fp_rhoF);
    fclose(fp_rhoG);

    int x, y, z, k;
    for(k=0; k<num_node; k++)
    {
        x = n_to_XYZ[0*num_node + k];
        y = n_to_XYZ[1*num_node + k];
        z = n_to_XYZ[2*num_node + k];
        simgpu_->rhoF_h[k] = rhoF[z*ny*nx + y*nx + x];
        simgpu_->rhoG_h[k] = rhoG[z*ny*nx + y*nx + x];
        simgpu_->vxF_h[k] = 0.00;
        simgpu_->vyF_h[k] = 0.00;
        simgpu_->vzF_h[k] = 0.00;
        simgpu_->vxG_h[k] = 0.00;
        simgpu_->vyG_h[k] = 0.00;
        simgpu_->vzG_h[k] = 0.00;
    }
    free(rhoF);
    free(rhoG);

    int i;
    for(k=0; k<num_node; k++)
        for(i=0;i<Q;i++) {
            simgpu_->f0_h[i*num_node + k] = w_h[i]*simgpu_->rhoF_h[k];
            simgpu_->g0_h[i*num_node + k] = w_h[i]*simgpu_->rhoG_h[k];
        }
}

void copyBackData(SIMGPU *simgpu_)
{
    hipMemcpy(simgpu_->f0_h, simgpu_->f0_d, simgpu_->num_node*Q*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->g0_h, simgpu_->g0_d, simgpu_->num_node*Q*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);

    hipMemcpy(simgpu_->rhoF_h, simgpu_->rhoF_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->rhoG_h, simgpu_->rhoG_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vxF_h, simgpu_->vxF_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vyF_h, simgpu_->vyF_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vzF_h, simgpu_->vzF_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vxG_h, simgpu_->vxG_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vyG_h, simgpu_->vyG_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
    hipMemcpy(simgpu_->vzG_h, simgpu_->vzG_d, simgpu_->num_node*sizeof(MY_DTYPE), hipMemcpyDeviceToHost);
}

void outputData(SIMGPU *simgpu_, int it)
{
    FILE *fp_rhoF, *fp_rhoG, *fp_vx, *fp_vy, *fp_vz;
    MY_DTYPE *rhoF, *rhoG, *vx, *vy, *vz;
    unsigned short *n_to_XYZ;
    int nz, ny, nx, num_node;
    int size;
    n_to_XYZ = simgpu_->n_to_XYZ;
    nz = simgpu_->nz;
    ny = simgpu_->ny;
    nx = simgpu_->nx;
    size = nx*ny*nz;
    num_node = simgpu_->num_node;
    rhoF = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    rhoG = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vx = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vy = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vz = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    if(!(rhoF && rhoG && vx && vy && vz)){
        fprintf(stderr, "Memory allocating error, simgpu.c\n");
        exit(-1);
    }

    int k;
    unsigned short x, y, z;
    /*MY_DTYPE rho_f, rho_g, vx_f, vy_f, vz_f, vx_g, vy_g, vz_g;*/
    for(k=0; k<num_node; k++)
    {
        x = n_to_XYZ[0*num_node + k];
        y = n_to_XYZ[1*num_node + k];
        z = n_to_XYZ[2*num_node + k];
        rhoF[z*nx*ny+y*nx+x] = simgpu_->rhoF_h[k];
        rhoG[z*nx*ny+y*nx+x] = simgpu_->rhoG_h[k];
        vx[z*nx*ny+y*nx+x] = simgpu_->vx_h[k];
        vy[z*nx*ny+y*nx+x] = simgpu_->vy_h[k];
        vz[z*nx*ny+y*nx+x] = simgpu_->vz_h[k];
    }

    char rhoF_filename[100];
    char rhoG_filename[100];
    sprintf(rhoF_filename, "rhoF_%08d.dat", it);
    sprintf(rhoG_filename, "rhoG_%08d.dat", it);

    /*fp_rhoF = fopen("rhoF.dat", "w");*/
    /*fp_rhoG = fopen("rhoG.dat", "w");*/
    fp_rhoF = fopen(rhoF_filename, "w");
    fp_rhoG = fopen(rhoG_filename, "w");

    fp_vx = fopen("vx.dat", "w");
    fp_vy = fopen("vy.dat", "w");
    fp_vz = fopen("vz.dat", "w");
    if(!(fp_rhoF&& fp_rhoG && fp_vx && fp_vy && fp_vz)){
        fprintf(stderr, "rho File openning error, simgpu.cu\n");
        exit(-1);
    }
    /*z = nz/2;*/
    x = nx/2;
    /*for(x=0; x<nx; x++)*/
    for(z=0; z<nz; z++)
    {
        for(y=0; y<ny; y++)
        {
            fprintf(fp_rhoF, "% 16.14e ", rhoF[z*nx*ny + y*nx + x]);
            fprintf(fp_rhoG, "% 16.14e ", rhoG[z*nx*ny + y*nx + x]);
            fprintf(fp_vx, "% 16.14e ", vx[z*nx*ny + y*nx + x]);
            fprintf(fp_vy, "% 16.14e ", vy[z*nx*ny + y*nx + x]);
            fprintf(fp_vz, "% 16.14e ", vz[z*nx*ny + y*nx + x]);
        }
        fprintf(fp_rhoF, "\n");
        fprintf(fp_rhoG, "\n");
        fprintf(fp_vx, "\n");
        fprintf(fp_vy, "\n");
        fprintf(fp_vz, "\n");
    }

    fclose(fp_rhoF);
    fclose(fp_rhoG);
    fclose(fp_vx);
    fclose(fp_vy);
    fclose(fp_vz);
    free(rhoF);
    free(rhoG);
    free(vx);
    free(vy);
    free(vz);
}

void saveTecplot(SIMGPU *simgpu_)
{

    int nx, ny, nz, size, num_node;
    unsigned short *n_to_XYZ;
    nx = simgpu_->nx;
    ny = simgpu_->ny;
    nz = simgpu_->nz;
    size = nx*ny*nz;
    num_node = simgpu_->num_node;
    n_to_XYZ = simgpu_->n_to_XYZ;

    MY_DTYPE rhoff, rhogg,  vxx, vyy, vzz;
    MY_DTYPE *rhoF, *rhoG, *vx, *vy, *vz;
    MY_DTYPE *flag;
    MY_DTYPE *X, *Y, *Z;
    int x, y, z, k ;

    rhoF = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    rhoG = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vx = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vy = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    vz = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    flag = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    X = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    Y = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));
    Z = (MY_DTYPE *)calloc(size,sizeof(MY_DTYPE));

    if(!(rhoF && rhoG && vx && vy && vz)){
        fprintf(stderr, "Memory allocating error, simgpu.c\n");
        exit(-1);
    }

    int zi, yi, xi;
    for(zi=0; zi<nz; zi++)
        for(yi=0; yi<ny; yi++)
            for(xi=0; xi<nx; xi++) {
                Z[zi*nx*ny+yi*nx+xi] = (MY_DTYPE)zi;
                Y[zi*nx*ny+yi*nx+xi] = (MY_DTYPE)yi;
                X[zi*nx*ny+yi*nx+xi] = (MY_DTYPE)xi;
                flag[zi*nx*ny+yi*nx+xi] = 1.0;
                rhoF[zi*nx*ny+yi*nx+xi] = -1.0;
                rhoG[zi*nx*ny+yi*nx+xi] = -1.0;
            }

    for(k=0; k<num_node; k++)
    {
        rhoff = simgpu_->rhoF_h[k];
        rhogg = simgpu_->rhoG_h[k];

        vxx = simgpu_->vx_h[k];
        vyy = simgpu_->vy_h[k];
        vzz = simgpu_->vz_h[k];

        x = n_to_XYZ[0*num_node + k];
        y = n_to_XYZ[1*num_node + k];
        z = n_to_XYZ[2*num_node + k];

        rhoF[z*nx*ny+y*nx+x] = rhoff;
        rhoG[z*nx*ny+y*nx+x] = rhogg;
        vx[z*nx*ny+y*nx+x] = vxx;
        vy[z*nx*ny+y*nx+x] = vyy;
        vz[z*nx*ny+y*nx+x] = vzz;
        flag[z*nx*ny+y*nx+x] = 0.0;
    }

    double SolTime;
    INTEGER4 Debug,I, III,DIsDouble,VIsDouble,IMax,JMax,KMax,ZoneType,StrandID,ParentZn,IsBlock;
    INTEGER4 ICellMax,JCellMax,KCellMax,NFConns,FNMode,ShrConn, FileType;

    Debug     = 0;
    VIsDouble = 0;
    DIsDouble = 0;
    IMax      = nx;
    JMax      = ny;
    KMax      = nz;
    ZoneType  = 0;      /* Ordered */
    SolTime   = 0.0;
    StrandID  = 0;     /* StaticZone */
    ParentZn  = 0;      /* No Parent */
    IsBlock   = 1;      /* Block */
    ICellMax  = 0;
    JCellMax  = 0;
    KCellMax  = 0;
    NFConns   = 0;
    FNMode    = 0;
    ShrConn   = 0;
    FileType  = 0;

    I = TECINI112("SIMPLE DATASET",
            "X Y Z rhoF rhoG vx vy vz flag",
            "tec.plt",
            ".",
            &FileType,
            &Debug,
            &VIsDouble);

    I = TECZNE112("Simple Zone",
            &ZoneType,
            &IMax,
            &JMax,
            &KMax,
            &ICellMax,
            &JCellMax,
            &KCellMax,
            &SolTime,
            &StrandID,
            &ParentZn,
            &IsBlock,
            &NFConns,
            &FNMode,
            0,              /* TotalNumFaceNodes */
            0,              /* NumConnectedBoundaryFaces */
            0,              /* TotalNumBoundaryConnections */
            NULL,           /* PassiveVarList */
            NULL,           /* ValueLocation = Nodal */
            NULL,           /* SharVarFromZone */
            &ShrConn);

    III = IMax*JMax*KMax;
    I = TECDAT112(&III,X,&DIsDouble);
    I = TECDAT112(&III,Y,&DIsDouble);
    I = TECDAT112(&III,Z,&DIsDouble);
    I = TECDAT112(&III,rhoF,&DIsDouble);
    I = TECDAT112(&III,rhoG,&DIsDouble);
    I = TECDAT112(&III,vx,&DIsDouble);
    I = TECDAT112(&III,vy,&DIsDouble);
    I = TECDAT112(&III,vz,&DIsDouble);
    I = TECDAT112(&III,flag,&DIsDouble);

    I = TECEND112();
}

void correctVel(SIMGPU *simgpu_)
{
    int num_node, k, i, I, ip;
    MY_DTYPE Fx_F, Fy_F, Fz_F;
    MY_DTYPE Fx_G, Fy_G, Fz_G;
    MY_DTYPE G_F, G_G;
    num_node = simgpu_->num_node;
    MY_DTYPE rhop_F, rhop_G;
    for(k=0; k<num_node; k++)
    {
        Fx_F = Fy_F = Fz_F = 0.0;
        Fx_G = Fy_G = Fz_G = 0.0;
        for(i=0; i<Q; i++)
        {
            I =simgpu_->node_map_h[i*num_node + k]; 
            ip = I/num_node;
            if(ip == i)
            {
                rhop_F = simgpu_->rhoG_h[k];
                rhop_G = simgpu_->rhoF_h[k];
                G_F = G_G = simgpu_->G_fg;
            }else
            {
                rhop_F = rhop_G = simgpu_->rhoS;
                G_F = simgpu_->G_fs;
                G_G = simgpu_->G_gs;
            }
            Fx_F += rhop_F*G_F*w_h[i]*e_h[i][0];
            Fy_F += rhop_F*G_F*w_h[i]*e_h[i][1];
            Fz_F += rhop_F*G_F*w_h[i]*e_h[i][2];
            Fx_G += rhop_G*G_G*w_h[i]*e_h[i][0];
            Fy_G += rhop_G*G_G*w_h[i]*e_h[i][1];
            Fz_G += rhop_G*G_G*w_h[i]*e_h[i][2];
        }

        Fx_F *= -simgpu_->rhoF_h[k]*18.0;
        Fy_F *= -simgpu_->rhoF_h[k]*18.0;
        Fz_F *= -simgpu_->rhoF_h[k]*18.0;
        Fx_G *= -simgpu_->rhoG_h[k]*18.0;
        Fy_G *= -simgpu_->rhoG_h[k]*18.0;
        Fz_G *= -simgpu_->rhoG_h[k]*18.0;

        simgpu_->vx_h[k] = (0.50*simgpu_->dt*Fx_F + simgpu_->vxF_h[k] 
                + 0.50*simgpu_->dt*Fx_G + simgpu_->vxG_h[k])/(
                    simgpu_->rhoF_h[k] + simgpu_->rhoG_h[k]);
        simgpu_->vy_h[k] = (0.50*simgpu_->dt*Fy_F + simgpu_->vyF_h[k] 
                + 0.50*simgpu_->dt*Fy_G + simgpu_->vyG_h[k])/(
                    simgpu_->rhoF_h[k] + simgpu_->rhoG_h[k]);
        simgpu_->vz_h[k] = (0.50*simgpu_->dt*Fz_F + simgpu_->vzF_h[k] 
                + 0.50*simgpu_->dt*Fz_G + simgpu_->vzG_h[k])/(
                    simgpu_->rhoF_h[k] + simgpu_->rhoG_h[k]);
    }
}

MY_DTYPE massError(SIMGPU *simgpu_)
{
    int num_node = simgpu_->num_node;
    int k;
    MY_DTYPE mass=0.0;
    for(k=0; k<num_node; k++)
        mass += simgpu_->rhoF_h[k] + simgpu_->rhoG_h[k];
    return ((mass-num_node)/num_node);
}

void saveLoadRecovery(int *it, SIMGPU *simgpu_, char saveOrLoad)
{
    FILE * fp;
    if(saveOrLoad == 's'){
        if((fp=fopen("recovery.bin", "wb")) == NULL)
        {
            fprintf(stderr, "Recovery data openning error\n");
            exit(-1);
        }

        fwrite(it, sizeof(int), 1, fp);
        if((fwrite(simgpu_->f0_h, sizeof(MY_DTYPE), Q*simgpu_->num_node, fp))!=Q*simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }

        if((fwrite(simgpu_->g0_h, sizeof(MY_DTYPE), Q*simgpu_->num_node, fp))!=Q*simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }

        if((fwrite(simgpu_->rhoF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->rhoG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vxF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vyF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vzF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vxG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vyG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
        if((fwrite(simgpu_->vzG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data writing error\n");
            exit(-1);
        }
    }
    else
    {
        if((fp=fopen("recovery.bin", "rb")) == NULL)
        {
            fprintf(stderr, "Recovery data openning error\n");
            exit(-1);
        }
        fread(it, sizeof(int), 1, fp);
        if((fread(simgpu_->f0_h, sizeof(MY_DTYPE), Q*simgpu_->num_node, fp))!=Q*simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->g0_h, sizeof(MY_DTYPE), Q*simgpu_->num_node, fp))!=Q*simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->rhoF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->rhoG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vxF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vyF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vzF_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vxG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vyG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
        if((fread(simgpu_->vzG_h, sizeof(MY_DTYPE), simgpu_->num_node, fp))!=simgpu_->num_node)
        {
            fprintf(stderr, "Recovery data reading error\n");
            exit(-1);
        }
    }
    fclose(fp);
}

__global__ void LBCollProp (
        int it,
        int num_node,
        MY_DTYPE omegaF, MY_DTYPE omegaG, MY_DTYPE rhoS,
        MY_DTYPE G_fg, MY_DTYPE G_fs, MY_DTYPE G_gs,
        MY_DTYPE *fInF, MY_DTYPE *fOutF, 
        MY_DTYPE *fInG, MY_DTYPE *fOutG,
        MY_DTYPE *rhoF, MY_DTYPE *rhoG,
        MY_DTYPE *vxF, MY_DTYPE *vxG,
        MY_DTYPE *vyF, MY_DTYPE *vyG,
        MY_DTYPE *vzF, MY_DTYPE *vzG,
        unsigned int *node_map)
{
    int k = blockIdx.y*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    MY_DTYPE rho_F, vx_F, vy_F, vz_F;
    MY_DTYPE rho_G, vx_G, vy_G, vz_G;
    MY_DTYPE Fx_F, Fy_F, Fz_F;
    MY_DTYPE Fx_G, Fy_G, Fz_G;
    MY_DTYPE F_F, F_G;
    MY_DTYPE vx_eq, vy_eq, vz_eq, vv_eq;
    int  kp;
    int I1,I2,I3,I4,I5,I6,I7,I8,I9,I10,I11,I12,I13,I14,I15,I16,I17,I18;
    MY_DTYPE Feq, rhoP_F, rhoP_G;
    MY_DTYPE G_F, G_G;
    int ip;

    if(k < num_node) //valid threads
    {
        rho_F = rhoF[k];
        rho_G = rhoG[k];

        vx_F = vxF[k];
        vy_F = vyF[k];
        vz_F = vzF[k];
        vx_G = vxG[k];
        vy_G = vyG[k];
        vz_G = vzG[k];

        //Optimiaze
        I1  = node_map[ 1*num_node+k]; 
        I2  = node_map[ 2*num_node+k];
        I3  = node_map[ 3*num_node+k];
        I4  = node_map[ 4*num_node+k];
        I5  = node_map[ 5*num_node+k];
        I6  = node_map[ 6*num_node+k];
        I7  = node_map[ 7*num_node+k];
        I8  = node_map[ 8*num_node+k];
        I9  = node_map[ 9*num_node+k];
        I10 = node_map[10*num_node+k];
        I11 = node_map[11*num_node+k];
        I12 = node_map[12*num_node+k];
        I13 = node_map[13*num_node+k];
        I14 = node_map[14*num_node+k];
        I15 = node_map[15*num_node+k];
        I16 = node_map[16*num_node+k];
        I17 = node_map[17*num_node+k];
        I18 = node_map[18*num_node+k];

        Fx_F = Fy_F = Fz_F = 0.00;
        Fx_G = Fy_G = Fz_G = 0.00;

        /////////////////////////////////////////////////////////////////////////////
        ip = I1/num_node;
        kp = I1-ip*num_node;
        if(ip == 1) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F =G_fs; G_G = G_gs; }
        Fx_F += w_d[ 1 ]*rhoP_F*G_F;
        Fx_G += w_d[ 1 ]*rhoP_G*G_G;

        ip = I2/num_node;
        kp = I2-ip*num_node;
        if(ip == 2) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F -= w_d[ 2 ]*rhoP_F*G_F;
        Fx_G -= w_d[ 2 ]*rhoP_G*G_G;

        ip = I3/num_node;
        kp = I3-ip*num_node;
        if(ip == 3) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F += w_d[ 3 ]*rhoP_F*G_F;
        Fy_G += w_d[ 3 ]*rhoP_G*G_G;

        ip = I4/num_node;
        kp = I4-ip*num_node;
        if(ip == 4) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F -= w_d[ 4 ]*rhoP_F*G_F;
        Fy_G -= w_d[ 4 ]*rhoP_G*G_G;

        ip = I5/num_node;
        kp = I5-ip*num_node;
        if(ip == 5) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fz_F += w_d[ 5 ]*rhoP_F*G_F;
        Fz_G += w_d[ 5 ]*rhoP_G*G_G;

        ip = I6/num_node;
        kp = I6-ip*num_node;
        if(ip == 6) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fz_F -= w_d[ 6 ]*rhoP_F*G_F;
        Fz_G -= w_d[ 6 ]*rhoP_G*G_G;

        ip = I7/num_node;
        kp = I7-ip*num_node;
        if(ip == 7) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F += w_d[ 7 ]*rhoP_F*G_F;
        Fy_G += w_d[ 7 ]*rhoP_G*G_G;
        Fz_F += w_d[ 7 ]*rhoP_F*G_F;
        Fz_G += w_d[ 7 ]*rhoP_G*G_G;

        ip = I8/num_node;
        kp = I8-ip*num_node;
        if(ip == 8) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F -= w_d[ 8 ]*rhoP_F*G_F;
        Fy_G -= w_d[ 8 ]*rhoP_G*G_G;
        Fz_F -= w_d[ 8 ]*rhoP_F*G_F;
        Fz_G -= w_d[ 8 ]*rhoP_G*G_G;

        ip = I9/num_node;
        kp = I9-ip*num_node;
        if(ip == 9) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F -= w_d[ 9 ]*rhoP_F*G_F;
        Fy_G -= w_d[ 9 ]*rhoP_G*G_G;
        Fz_F += w_d[ 9 ]*rhoP_F*G_F;
        Fz_G += w_d[ 9 ]*rhoP_G*G_G;

        ip = I10/num_node;
        kp = I10-ip*num_node;
        if(ip == 10) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fy_F += w_d[ 10 ]*rhoP_F*G_F;
        Fy_G += w_d[ 10 ]*rhoP_G*G_G;
        Fz_F -= w_d[ 10 ]*rhoP_F*G_F;
        Fz_G -= w_d[ 10 ]*rhoP_G*G_G;

        ip = I11/num_node;
        kp = I11-ip*num_node;
        if(ip == 11) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F -= w_d[ 11 ]*rhoP_F*G_F;
        Fx_G -= w_d[ 11 ]*rhoP_G*G_G;
        Fz_F -= w_d[ 11 ]*rhoP_F*G_F;
        Fz_G -= w_d[ 11 ]*rhoP_G*G_G;

        ip = I12/num_node;
        kp = I12-ip*num_node;
        if(ip == 12) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F += w_d[ 12 ]*rhoP_F*G_F;
        Fx_G += w_d[ 12 ]*rhoP_G*G_G;
        Fz_F += w_d[ 12 ]*rhoP_F*G_F;
        Fz_G += w_d[ 12 ]*rhoP_G*G_G;

        ip = I13/num_node;
        kp = I13-ip*num_node;
        if(ip == 13) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F -= w_d[ 13 ]*rhoP_F*G_F;
        Fx_G -= w_d[ 13 ]*rhoP_G*G_G;
        Fz_F += w_d[ 13 ]*rhoP_F*G_F;
        Fz_G += w_d[ 13 ]*rhoP_G*G_G;

        ip = I14/num_node;
        kp = I14-ip*num_node;
        if(ip == 14) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F += w_d[ 14 ]*rhoP_F*G_F;
        Fx_G += w_d[ 14 ]*rhoP_G*G_G;
        Fz_F -= w_d[ 14 ]*rhoP_F*G_F;
        Fz_G -= w_d[ 14 ]*rhoP_G*G_G;

        ip = I15/num_node;
        kp = I15-ip*num_node;
        if(ip == 15) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F -= w_d[ 15 ]*rhoP_F*G_F;
        Fx_G -= w_d[ 15 ]*rhoP_G*G_G;
        Fy_F += w_d[ 15 ]*rhoP_F*G_F;
        Fy_G += w_d[ 15 ]*rhoP_G*G_G;

        ip = I16/num_node;
        kp = I16-ip*num_node;
        if(ip == 16) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F += w_d[ 16 ]*rhoP_F*G_F;
        Fx_G += w_d[ 16 ]*rhoP_G*G_G;
        Fy_F -= w_d[ 16 ]*rhoP_F*G_F;
        Fy_G -= w_d[ 16 ]*rhoP_G*G_G;

        ip = I17/num_node;
        kp = I17-ip*num_node;
        if(ip == 17) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F -= w_d[ 17 ]*rhoP_F*G_F;
        Fx_G -= w_d[ 17 ]*rhoP_G*G_G;
        Fy_F -= w_d[ 17 ]*rhoP_F*G_F;
        Fy_G -= w_d[ 17 ]*rhoP_G*G_G;

        ip = I18/num_node;
        kp = I18-ip*num_node;
        if(ip == 18) { rhoP_F = rhoG[kp]; rhoP_G = rhoF[kp]; G_F = G_G = G_fg;
        } else{ rhoP_F = rhoP_G = rhoS; G_F = G_fs; G_G = G_gs; }
        Fx_F += w_d[ 18 ]*rhoP_F*G_F;
        Fx_G += w_d[ 18 ]*rhoP_G*G_G;
        Fy_F += w_d[ 18 ]*rhoP_F*G_F;
        Fy_G += w_d[ 18 ]*rhoP_G*G_G;
        /////////////////////////////////////////////////////////////////////////////

        Fx_F *=-18.00;
        Fy_F *=-18.00;
        Fz_F *=-18.00;
        Fx_G *=-18.00;
        Fy_G *=-18.00;
        Fz_G *=-18.00;

        vx_F += 0.50*Fx_F*rho_F;
        vy_F += 0.50*Fy_F*rho_F;
        vz_F += 0.50*Fz_F*rho_F;
        vx_G += 0.50*Fx_G*rho_G;
        vy_G += 0.50*Fy_G*rho_G;
        vz_G += 0.50*Fz_G*rho_G;

        vx_eq = (vx_F*omegaF + vx_G*omegaG)/(rho_F*omegaF + rho_G*omegaG);
        vy_eq = (vy_F*omegaF + vy_G*omegaG)/(rho_F*omegaF + rho_G*omegaG);
        vz_eq = (vz_F*omegaF + vz_G*omegaG)/(rho_F*omegaF + rho_G*omegaG);
        vv_eq = 1.50*(vx_eq*vx_eq + vy_eq*vy_eq + vz_eq*vz_eq);

        if(it==0)
        {
            vx_eq = 0.00;
            vy_eq = 0.00;
            vz_eq = 0.00;
            vv_eq = 0.00;
        }else{
            vxF[k] = vx_F;
            vyF[k] = vy_F;
            vzF[k] = vz_F;
            vxG[k] = vx_G;
            vyG[k] = vy_G;
            vzG[k] = vz_G;
        }

        //Force
        Fz_F += 1.0e-5;
        Fz_G += 1.0e-5;

        Fx_F *= 3*(1.00 - 0.50*omegaF);
        Fy_F *= 3*(1.00 - 0.50*omegaF);
        Fz_F *= 3*(1.00 - 0.50*omegaF);
        Fx_G *= 3*(1.00 - 0.50*omegaG);
        Fy_G *= 3*(1.00 - 0.50*omegaG);
        Fz_G *= 3*(1.00 - 0.50*omegaG);

        //Optimiaze
        /*Feq = 0.0;*/
        //end Optimiaze

        ////////////////////////////////////////////////////////////////////////////
        F_F = Fx_F*(e_d[0][0] - vx_eq) + Fy_F*(e_d[0][1] - vy_eq) +Fz_F*(e_d[0][2] - vz_eq);
        F_G = Fx_G*(e_d[0][0] - vx_eq) + Fy_G*(e_d[0][1] - vy_eq) +Fz_G*(e_d[0][2] - vz_eq);
        Feq = FEQ_0(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[k ] = (1.0 - omegaF)*fInF[k+0*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[k ] = (1.0 - omegaG)*fInG[k+0*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[1][0] - vx_eq) + Fy_F*(e_d[1][1] - vy_eq) +Fz_F*(e_d[1][2] - vz_eq);
        F_G = Fx_G*(e_d[1][0] - vx_eq) + Fy_G*(e_d[1][1] - vy_eq) +Fz_G*(e_d[1][2] - vz_eq);
        Feq = FEQ_1(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I1] = (1.0 - omegaF)*fInF[k+1*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I1] = (1.0 - omegaG)*fInG[k+1*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[2][0] - vx_eq) + Fy_F*(e_d[2][1] - vy_eq) +Fz_F*(e_d[2][2] - vz_eq);
        F_G = Fx_G*(e_d[2][0] - vx_eq) + Fy_G*(e_d[2][1] - vy_eq) +Fz_G*(e_d[2][2] - vz_eq);
        Feq = FEQ_2(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I2] = (1.0 - omegaF)*fInF[k+2*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I2] = (1.0 - omegaG)*fInG[k+2*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[3][0] - vx_eq) + Fy_F*(e_d[3][1] - vy_eq) +Fz_F*(e_d[3][2] - vz_eq);
        F_G = Fx_G*(e_d[3][0] - vx_eq) + Fy_G*(e_d[3][1] - vy_eq) +Fz_G*(e_d[3][2] - vz_eq);
        Feq = FEQ_3(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I3] = (1.0 - omegaF)*fInF[k+3*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I3] = (1.0 - omegaG)*fInG[k+3*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[4][0] - vx_eq) + Fy_F*(e_d[4][1] - vy_eq) +Fz_F*(e_d[4][2] - vz_eq);
        F_G = Fx_G*(e_d[4][0] - vx_eq) + Fy_G*(e_d[4][1] - vy_eq) +Fz_G*(e_d[4][2] - vz_eq);
        Feq = FEQ_4(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I4] = (1.0 - omegaF)*fInF[k+4*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I4] = (1.0 - omegaG)*fInG[k+4*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[5][0] - vx_eq) + Fy_F*(e_d[5][1] - vy_eq) +Fz_F*(e_d[5][2] - vz_eq);
        F_G = Fx_G*(e_d[5][0] - vx_eq) + Fy_G*(e_d[5][1] - vy_eq) +Fz_G*(e_d[5][2] - vz_eq);
        Feq = FEQ_5(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I5] = (1.0 - omegaF)*fInF[k+5*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I5] = (1.0 - omegaG)*fInG[k+5*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[6][0] - vx_eq) + Fy_F*(e_d[6][1] - vy_eq) +Fz_F*(e_d[6][2] - vz_eq);
        F_G = Fx_G*(e_d[6][0] - vx_eq) + Fy_G*(e_d[6][1] - vy_eq) +Fz_G*(e_d[6][2] - vz_eq);
        Feq = FEQ_6(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I6] = (1.0 - omegaF)*fInF[k+6*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I6] = (1.0 - omegaG)*fInG[k+6*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[7][0] - vx_eq) + Fy_F*(e_d[7][1] - vy_eq) +Fz_F*(e_d[7][2] - vz_eq);
        F_G = Fx_G*(e_d[7][0] - vx_eq) + Fy_G*(e_d[7][1] - vy_eq) +Fz_G*(e_d[7][2] - vz_eq);
        Feq = FEQ_7(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I7] = (1.0 - omegaF)*fInF[k+7*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I7] = (1.0 - omegaG)*fInG[k+7*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[8][0] - vx_eq) + Fy_F*(e_d[8][1] - vy_eq) +Fz_F*(e_d[8][2] - vz_eq);
        F_G = Fx_G*(e_d[8][0] - vx_eq) + Fy_G*(e_d[8][1] - vy_eq) +Fz_G*(e_d[8][2] - vz_eq);
        Feq = FEQ_8(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I8] = (1.0 - omegaF)*fInF[k+8*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I8] = (1.0 - omegaG)*fInG[k+8*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[9][0] - vx_eq) + Fy_F*(e_d[9][1] - vy_eq) +Fz_F*(e_d[9][2] - vz_eq);
        F_G = Fx_G*(e_d[9][0] - vx_eq) + Fy_G*(e_d[9][1] - vy_eq) +Fz_G*(e_d[9][2] - vz_eq);
        Feq = FEQ_9(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I9] = (1.0 - omegaF)*fInF[k+9*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I9] = (1.0 - omegaG)*fInG[k+9*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[10][0] - vx_eq) + Fy_F*(e_d[10][1] - vy_eq) +Fz_F*(e_d[10][2] - vz_eq);
        F_G = Fx_G*(e_d[10][0] - vx_eq) + Fy_G*(e_d[10][1] - vy_eq) +Fz_G*(e_d[10][2] - vz_eq);
        Feq = FEQ_10(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I10] = (1.0 - omegaF)*fInF[k+10*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I10] = (1.0 - omegaG)*fInG[k+10*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[11][0] - vx_eq) + Fy_F*(e_d[11][1] - vy_eq) +Fz_F*(e_d[11][2] - vz_eq);
        F_G = Fx_G*(e_d[11][0] - vx_eq) + Fy_G*(e_d[11][1] - vy_eq) +Fz_G*(e_d[11][2] - vz_eq);
        Feq = FEQ_11(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I11] = (1.0 - omegaF)*fInF[k+11*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I11] = (1.0 - omegaG)*fInG[k+11*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[12][0] - vx_eq) + Fy_F*(e_d[12][1] - vy_eq) +Fz_F*(e_d[12][2] - vz_eq);
        F_G = Fx_G*(e_d[12][0] - vx_eq) + Fy_G*(e_d[12][1] - vy_eq) +Fz_G*(e_d[12][2] - vz_eq);
        Feq = FEQ_12(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I12] = (1.0 - omegaF)*fInF[k+12*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I12] = (1.0 - omegaG)*fInG[k+12*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[13][0] - vx_eq) + Fy_F*(e_d[13][1] - vy_eq) +Fz_F*(e_d[13][2] - vz_eq);
        F_G = Fx_G*(e_d[13][0] - vx_eq) + Fy_G*(e_d[13][1] - vy_eq) +Fz_G*(e_d[13][2] - vz_eq);
        Feq = FEQ_13(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I13] = (1.0 - omegaF)*fInF[k+13*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I13] = (1.0 - omegaG)*fInG[k+13*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[14][0] - vx_eq) + Fy_F*(e_d[14][1] - vy_eq) +Fz_F*(e_d[14][2] - vz_eq);
        F_G = Fx_G*(e_d[14][0] - vx_eq) + Fy_G*(e_d[14][1] - vy_eq) +Fz_G*(e_d[14][2] - vz_eq);
        Feq = FEQ_14(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I14] = (1.0 - omegaF)*fInF[k+14*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I14] = (1.0 - omegaG)*fInG[k+14*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[15][0] - vx_eq) + Fy_F*(e_d[15][1] - vy_eq) +Fz_F*(e_d[15][2] - vz_eq);
        F_G = Fx_G*(e_d[15][0] - vx_eq) + Fy_G*(e_d[15][1] - vy_eq) +Fz_G*(e_d[15][2] - vz_eq);
        Feq = FEQ_15(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I15] = (1.0 - omegaF)*fInF[k+15*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I15] = (1.0 - omegaG)*fInG[k+15*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[16][0] - vx_eq) + Fy_F*(e_d[16][1] - vy_eq) +Fz_F*(e_d[16][2] - vz_eq);
        F_G = Fx_G*(e_d[16][0] - vx_eq) + Fy_G*(e_d[16][1] - vy_eq) +Fz_G*(e_d[16][2] - vz_eq);
        Feq = FEQ_16(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I16] = (1.0 - omegaF)*fInF[k+16*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I16] = (1.0 - omegaG)*fInG[k+16*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[17][0] - vx_eq) + Fy_F*(e_d[17][1] - vy_eq) +Fz_F*(e_d[17][2] - vz_eq);
        F_G = Fx_G*(e_d[17][0] - vx_eq) + Fy_G*(e_d[17][1] - vy_eq) +Fz_G*(e_d[17][2] - vz_eq);
        Feq = FEQ_17(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I17] = (1.0 - omegaF)*fInF[k+17*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I17] = (1.0 - omegaG)*fInG[k+17*num_node] + (omegaG + F_G)*Feq*rho_G;

        F_F = Fx_F*(e_d[18][0] - vx_eq) + Fy_F*(e_d[18][1] - vy_eq) +Fz_F*(e_d[18][2] - vz_eq);
        F_G = Fx_G*(e_d[18][0] - vx_eq) + Fy_G*(e_d[18][1] - vy_eq) +Fz_G*(e_d[18][2] - vz_eq);
        Feq = FEQ_18(vx_eq, vy_eq, vz_eq, vv_eq);
        fOutF[I18] = (1.0 - omegaF)*fInF[k+18*num_node] + (omegaF + F_F)*Feq*rho_F;
        fOutG[I18] = (1.0 - omegaG)*fInG[k+18*num_node] + (omegaG + F_G)*Feq*rho_G;
    }
}


__global__ void LBUpdateMacro(
        int num_node,
        MY_DTYPE *rhoF, MY_DTYPE *rhoG,
        MY_DTYPE *vxF, MY_DTYPE *vxG,
        MY_DTYPE *vyF, MY_DTYPE *vyG,
        MY_DTYPE *vzF, MY_DTYPE *vzG,
        MY_DTYPE *fInF, MY_DTYPE *fInG)
{
    int k = blockIdx.y*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    MY_DTYPE fi, rho, vx, vy, vz;
    int i;
    if(k < num_node) //valid threads
    {

        rho = vx = vy = vz = 0.00;
#pragma unroll
        for(i=0; i<Q; i++){
            fi = fInF[i*num_node + k];
            rho += fi;
            vx += fi*e_d[i][0];
            vy += fi*e_d[i][1];
            vz += fi*e_d[i][2];
        }
        rhoF[k] = rho;
        vxF[k] = vx;
        vyF[k] = vy;
        vzF[k] = vz;

        rho = vx = vy = vz = 0.00;
#pragma unroll
        for(i=0; i<Q; i++){
            fi = fInG[i*num_node + k];
            rho += fi;
            vx += fi*e_d[i][0];
            vy += fi*e_d[i][1];
            vz += fi*e_d[i][2];
        }
        rhoG[k] = rho;
        vxG[k] = vx;
        vyG[k] = vy;
        vzG[k] = vz;
    }
}
